/* Copyright (c) 2016-2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * Performance note: Typically the color space conversions take up a
 * negligible amount of run time. The following kernels have therefore
 * not been optimized.
 */

#include <cassert>
#include <cstddef>
#include <cinttypes>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>

static inline __device__ float
clamp(const float v, const float low, const float high) {
    return v < low ? low : v > high ? high : v;
}

static inline __device__ float
rgb2y(const uint8_t r, const uint8_t g, const uint8_t b) {
    return 0.299f*(float)r + 0.587f*(float)g + 0.114f*(float)b;
}
static inline __device__ float
rgb2u(const uint8_t r, const uint8_t g, const uint8_t b) {
    const float y = rgb2y(r,g,b);
    return clamp(-(((-(float)b + y) / 1.732446f) - 128.f), 0.f, 255.f);
}
static inline __device__ float
rgb2v(const uint8_t r, const uint8_t g, const uint8_t b) {
    const float y = rgb2y(r,g,b);
    const float u = rgb2u(r,g,b);
    return clamp((y - (0.337633f*(u-128.f)) - (float)g) / 0.698001f + 128.f,
                 0.f, 255.f);
}

/* Converts from RGB data to NV12.  NV12's "U" and "V" channels are interleaved
 * and subsampled 2x2.  Note the RGB data are not pitched. */
extern "C" __global__ void
rgb2yuv(const uint8_t* const __restrict rgb,
        const uint32_t width, const uint32_t height,
        const uint32_t widthUser, const uint32_t heightUser, const uint32_t c/*omponents*/,
        uint8_t* const __restrict yuv, unsigned pitch) {
    const uint32_t x = blockIdx.x*blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y*blockDim.y + threadIdx.y;
    const uint32_t i = y*pitch + x;

    if(x >= width || y >= height || i >= pitch*height)
        return;

    /* Repeat edge pixels for padded areas */
    const uint32_t _x = min(x, widthUser - 1);
    const uint32_t _y = min(y, heightUser - 1);
    const uint32_t j = _y * widthUser + _x;

    assert(pitch >= width);
    assert(i < pitch * height);
    assert(j < widthUser * heightUser);
    assert(width <= 4096);
    assert(height <= 4096);
    assert(c == 3 || c == 4);
    assert(pitch <= 4096);

    uint8_t* __restrict Y = yuv;
    Y[i] = (uint8_t)clamp(rgb2y(rgb[j*c+0], rgb[j*c+1], rgb[j*c+2]), 0, 255);

    /* U+V are downsampled 2x per dimension.  So kill off 3 of every 4 threads
     * that reach here; only one will do the writes into U and V. */
    /* thought: use x0 to write into U and x1 to write into V, to spread load? */
    if(x&1 == 1 || y&1 == 1) {
        return;
    }
    uint8_t* __restrict uv = yuv + pitch*height;
    const uint32_t uvidx = y/2*(pitch/2) + x/2;

    const uint32_t idx[4] = {
        min((_y+0)*widthUser + _x+0, widthUser*heightUser - 1),
        min((_y+0)*widthUser + _x+1, widthUser*heightUser - 1),
        min((_y+1)*widthUser + _x+0, widthUser*heightUser - 1),
        min((_y+1)*widthUser + _x+1, widthUser*heightUser - 1),
    };
    const float u[4] = {
        rgb2u(rgb[idx[0]*c+0], rgb[idx[0]*c+1], rgb[idx[0]*c+2]),
        rgb2u(rgb[idx[1]*c+0], rgb[idx[1]*c+1], rgb[idx[1]*c+2]),
        rgb2u(rgb[idx[2]*c+0], rgb[idx[2]*c+1], rgb[idx[2]*c+2]),
        rgb2u(rgb[idx[3]*c+0], rgb[idx[3]*c+1], rgb[idx[3]*c+2])
    };
    const float v[4] = {
        rgb2v(rgb[idx[0]*c+0], rgb[idx[0]*c+1], rgb[idx[0]*c+2]),
        rgb2v(rgb[idx[1]*c+0], rgb[idx[1]*c+1], rgb[idx[1]*c+2]),
        rgb2v(rgb[idx[2]*c+0], rgb[idx[2]*c+1], rgb[idx[2]*c+2]),
        rgb2v(rgb[idx[3]*c+0], rgb[idx[3]*c+1], rgb[idx[3]*c+2])
    };
    uv[uvidx*2+0] = (uint8_t)clamp((u[0] + u[1] + u[2] + u[3]) / 4.0, 0, 255);
    uv[uvidx*2+1] = (uint8_t)clamp((v[0] + v[1] + v[2] + v[3]) / 4.0, 0, 255);
}

static inline __device__ float
yuv2r(const uint8_t y, const uint8_t u, const uint8_t v) {
    (void)u;
    return (y-16)*1.164f + (1.596f * (v-128));
}
static inline __device__ float
yuv2g(const uint8_t y, const uint8_t u, const uint8_t v) {
    return (y-16)*1.164f + (u-128)*-0.392f + (v-128)*-0.813f;
}
static inline __device__ float
yuv2b(const uint8_t y, const uint8_t u, const uint8_t v) {
    (void)v;
    return (y-16)*1.164f + (u-128)*2.017f;
}

/* Convert back from NV12 to RGB.   Note the RGB buffer is not pitched. */
extern "C" __global__ void
yuv2rgb(const uint8_t* const __restrict yuv,
        const uint32_t width, const uint32_t height,
        uint32_t widthUser, uint32_t heightUser, unsigned pitch,
        uint8_t* const __restrict rgb) {
    const uint32_t x = blockIdx.x*blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y*blockDim.y + threadIdx.y;
    const uint32_t i = y*pitch + x;
    const uint32_t j = y*widthUser + x;
    if(x >= widthUser || y >= heightUser || x >= width || y >= height || i >= pitch*height || j >= widthUser * heightUser) {
        return;
    }
    assert(i < pitch*height);
    assert(j < widthUser * heightUser);
    assert(width <= 4096);
    assert(height <= 4096);
    assert(pitch <= 4096);
    const uint8_t* __restrict Y = yuv;
    const uint8_t* __restrict uv = yuv + pitch*height;
    const uint32_t xx = min(x+1, width-1);
    const uint32_t yy = min(y+1, height-1);
    const uint32_t idx[4] = {
        y/2*pitch/2 + x/2,
        y/2*pitch/2 + xx/2,
        yy/2*pitch/2 + x/2,
        yy/2*pitch/2 + xx/2,
    };
    const uint8_t u[4] = {
        uv[idx[0]*2+0], uv[idx[1]*2+0], uv[idx[2]*2+0], uv[idx[3]*2+0]
    };
    const uint8_t v[4] = {
        uv[idx[0]*2+1], uv[idx[1]*2+1], uv[idx[2]*2+1], uv[idx[3]*2+1]
    };
    rgb[j*3+0] = clamp(
                (yuv2r(Y[i], u[0], v[0]) + yuv2r(Y[i], u[1], v[1]) +
            yuv2r(Y[i], u[2], v[2]) + yuv2r(Y[i], u[3], v[3])) / 4.0, 0, 255
            );
    rgb[j*3+1] = clamp(
                (yuv2g(Y[i], u[0], v[0]) + yuv2g(Y[i], u[1], v[1]) +
            yuv2g(Y[i], u[2], v[2]) + yuv2g(Y[i], u[3], v[3])) / 4.0, 0, 255
            );
    rgb[j*3+2] = clamp(
                (yuv2b(Y[i], u[0], v[0]) + yuv2b(Y[i], u[1], v[1]) +
            yuv2b(Y[i], u[2], v[2]) + yuv2b(Y[i], u[3], v[3])) / 4.0, 0, 255
            );
}

extern "C" hipError_t
launch_yuv2rgb(hipDeviceptr_t nv12, uint32_t width, uint32_t height,
               uint32_t widthUser, uint32_t heightUser, unsigned pitch,
               hipDeviceptr_t rgb, hipStream_t strm) {
    /* NvCodec maxes out at 8k anyway. */
    assert(width <= 8192);
    assert(height <= 8192);
    /* NvCodec can't give us a height that isn't evenly divisible. */
    assert(height%2 == 0);
    const void* args[] = {
        (void*)&nv12, &width, &height, &widthUser, &heightUser, &pitch, (void*)&rgb, 0
    };
    const dim3 gdim = {(unsigned)(width/16)+1, (unsigned)(height/2), 1};
    const dim3 bdim = {16, 2, 1};
    const size_t shmem = 0;
    return hipLaunchKernel(reinterpret_cast<const void*>((const void**)yuv2rgb), gdim, bdim, (void**)args,
                            shmem, strm);
}

extern "C" hipError_t
launch_rgb2yuv(hipDeviceptr_t rgb, uint32_t width, uint32_t height,
               uint32_t widthUser, uint32_t heightUser, uint32_t ncomp,
               hipDeviceptr_t nv12, unsigned pitch, hipStream_t strm) {
    /* NvCodec maxes out at 8k anyway. */
    assert(width <= 8192);
    assert(height <= 8192);
    /* We only support RGB and RGBA data. */
    assert(ncomp == 3 || ncomp == 4);
    /* NvCodec can't give us a height that isn't evenly divisible. */
    assert(height%2 == 0);

    const void* args[] = {
        (void*)&rgb, &width, &height, &widthUser, &heightUser, &ncomp, (void*)&nv12, &pitch,
    };
    dim3 gdim = {(unsigned)(width/16)+1, (unsigned)(height/2), 1};
    dim3 bdim = {16, 2, 1};
    const size_t shmem = 0;
    return hipLaunchKernel(reinterpret_cast<const void*>((const void*)rgb2yuv), gdim, bdim, (void**)args,
                            shmem, strm);
}
