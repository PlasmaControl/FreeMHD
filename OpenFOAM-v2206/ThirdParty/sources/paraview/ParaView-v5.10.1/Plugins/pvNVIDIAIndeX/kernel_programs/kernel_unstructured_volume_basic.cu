#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright 2021 NVIDIA Corporation. All rights reserved.
 *****************************************************************************/

// # RTC Kernel:
// ** Structured Volume Basic **

// # Summary:
// A simple volume kernel for unstructure volumes.
// It samples the volume scalar value for the current position and
// maps it to color through the colormap. Finally allows the users to change
// the contrast of the volume on the fly through the "pfloat 1" GUI parameter.

NV_IDX_XAC_VERSION_1_0

// DON'T CHANGE THIS STRUCT !!!
// It maps the GUI parameters from the Custom Visual Element GUI to this kernel.
// floats maps {pfloat1, pfloat2, pfloat3, pfloat4} GUI Parameters.
// ints maps {pint1, pint2, pint3, pint4} GUI Parameters.
struct Custom_params
{
    float4 floats;  // floats array
    int4 ints;      // ints array
};

using namespace nv::index::xac;
using namespace nv::index::xaclib;

class Volume_sample_program
{
    NV_IDX_VOLUME_SAMPLE_PROGRAM

    const Custom_params*  m_custom_params;

public:
    NV_IDX_DEVICE_INLINE_MEMBER
    void initialize()
    {
        // maps Custom Visual Element GUI parameters to this kernel.
        m_custom_params = state.bind_parameter_buffer<Custom_params>(0);
    }

    NV_IDX_DEVICE_INLINE_MEMBER
    int execute(
        const Sample_info_self& sample_info,
              Sample_output&    sample_output) const
    {
        // maps "pfloat 1" GUI parameter as gamma correction
        float screen_gamma = (m_custom_params->floats.x/100.f + 1.f)/2.f;
        if(screen_gamma < 0.001f)
          screen_gamma = 0.001f;

        // sample volume
        const auto& cell_info = sample_info.sample_cell_info;
        float sample_value = sample_value = state.self.fetch_attribute<float>(0u, cell_info);

        // get colormap
        const auto colormap = state.self.get_colormap();

        // lookup color value
        sample_output.color = colormap.lookup(sample_value);

        // apply gamma correction
        sample_output.color = gamma_correct(sample_output.color, screen_gamma);

        return NV_IDX_PROG_OK;
    }
}; // class Volume_sample_program
