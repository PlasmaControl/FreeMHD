#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright 2021 NVIDIA Corporation. All rights reserved.
 *****************************************************************************/

// # RTC Kernel:
// ** Volume red **

// # Summary:
// A very simple volume kernel that creates an red volume full opaque.

NV_IDX_XAC_VERSION_1_0

// DON'T CHANGE THIS STRUCT !!!
// It maps the GUI parameters from the Custom Visual Element GUI to this kernel.
// floats maps {pfloat1, pfloat2, pfloat3, pfloat4} GUI Parameters.
// ints maps {pint1, pint2, pint3, pint4} GUI Parameters.
struct Custom_params
{
    float4 floats;  // floats array
    int4 ints;      // ints array
};

using namespace nv::index::xac;
using namespace nv::index::xaclib;

class Volume_sample_program
{
    NV_IDX_VOLUME_SAMPLE_PROGRAM

    const Custom_params*  m_custom_params;

public:
    NV_IDX_DEVICE_INLINE_MEMBER
    void initialize()
    {
        // maps Custom Visual Element GUI parameters to this kernel.
        m_custom_params = state.bind_parameter_buffer<Custom_params>(0);
    }

    NV_IDX_DEVICE_INLINE_MEMBER
    int execute(
        const Sample_info_self&  sample_info,
              Sample_output&     sample_output)
    {
        // Set the current volume sample to be red full opaque.
        sample_output.color = make_float4(1.f, 0.f, 0.f, 1.f);

        return NV_IDX_PROG_OK;
    }
}; // class Volume_sample_program
