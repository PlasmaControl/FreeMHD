#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright 2021 NVIDIA Corporation. All rights reserved.
 *****************************************************************************/

// # RTC Kernel:
// ** Structured volume opacity **

// # Summary:
// The same as "kernel_structured_volume_basic.cu" but it allows to
// change the opacity of the volume on the fly through the "pfloat 1" GUI parameter.

NV_IDX_XAC_VERSION_1_0

// DON'T CHANGE THIS STRUCT !!!
// It maps the GUI parameters from the Custom Visual Element GUI to this kernel.
// floats maps {pfloat1, pfloat2, pfloat3, pfloat4} GUI Parameters.
// ints maps {pint1, pint2, pint3, pint4} GUI Parameters.
struct Custom_params
{
    float4 floats;  // floats array
    int4 ints;      // ints array
};

using namespace nv::index;
using namespace nv::index::xac;

class Volume_sample_program
{
    NV_IDX_VOLUME_SAMPLE_PROGRAM

    const Custom_params*  m_custom_params;

public:
    NV_IDX_DEVICE_INLINE_MEMBER
    void initialize()
    {
        // maps Custom Visual Element GUI parameters to this kernel.
        m_custom_params = state.bind_parameter_buffer<Custom_params>(0);
    }

    NV_IDX_DEVICE_INLINE_MEMBER
    int execute(
        const Sample_info_self&  sample_info,
              Sample_output&     sample_output)
    {
        // map "pfloat 1" GUI parameter to opacity
        float opacity_boost = m_custom_params->floats.x/100.f;

        // get current sample position
        const float3& sample_position = sample_info.sample_position_object_space;
        const auto& sample_context = sample_info.sample_context;

        // get reference to the sparse volume
        const auto& sparse_volume = state.self;

        // generate a volume sampler and lookup sample a the current position
        const uint field_index = 0u;
        const auto sampler = sparse_volume.generate_sampler<float,
                                                         Volume_filter_mode::TRILINEAR,
                                                         Volume_classification_mode::POST_INTERPOLATION>(
                                                         field_index,
                                                         sample_context);

        const float sample_value = sampler.fetch_sample(sample_position);

        // Map sample to color
        const Colormap colormap = sparse_volume.get_colormap();
        float4 sample_color = colormap.lookup(sample_value);

        // apply opacity boost
        sample_color.w += opacity_boost;
        if(sample_color.w < 0.f)
          sample_color.w = 0.f;
        else if(sample_color.w > 1.f)
          sample_color.w = 1.f;

        // store the output color
        sample_output.set_color(sample_color);

        return NV_IDX_PROG_OK;
    }
}; // class Volume_sample_program
