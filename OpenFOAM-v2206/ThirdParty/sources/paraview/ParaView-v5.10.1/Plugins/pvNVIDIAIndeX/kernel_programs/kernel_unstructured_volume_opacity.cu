#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright 2021 NVIDIA Corporation. All rights reserved.
 *****************************************************************************/

// # RTC Kernel:
// ** Unstructured Volume Opacity **

// # Summary:
// Renders the unstructured volume but it allows the user to modify the
// opcity of the volume on the fly through the "pfloat 1" GUI parameter.

NV_IDX_XAC_VERSION_1_0

// DON'T CHANGE THIS STRUCT !!!
// It maps the GUI parameters from the Custom Visual Element GUI to this kernel.
// floats maps {pfloat1, pfloat2, pfloat3, pfloat4} GUI Parameters.
// ints maps {pint1, pint2, pint3, pint4} GUI Parameters.
struct Custom_params
{
    float4 floats;  // floats array
    int4 ints;      // ints array
};

using namespace nv::index::xac;
using namespace nv::index::xaclib;

class Volume_sample_program
{
    NV_IDX_VOLUME_SAMPLE_PROGRAM

    const Custom_params*  m_custom_params;

public:
    NV_IDX_DEVICE_INLINE_MEMBER
    void initialize()
    {
        // maps Custom Visual Element GUI parameters to this kernel.
        m_custom_params = state.bind_parameter_buffer<Custom_params>(0);
    }

    NV_IDX_DEVICE_INLINE_MEMBER
    int execute(
        const Sample_info_self& sample_info,
              Sample_output&    sample_output) const
    {
        // map "pfloat 1" GUI parameter to opacity boost
        float opacity_boost = m_custom_params->floats.x/100.f;

        // sample volume
        const auto& cell_info = sample_info.sample_cell_info;
        float sample_value = sample_value = state.self.fetch_attribute<float>(0u, cell_info);

        // get colormap
        const auto colormap = state.self.get_colormap();

        // lookup color value
        float4 sample_color = colormap.lookup(sample_value);

        // apply opacity boost
        sample_color.w += opacity_boost;
        if(sample_color.w < 0.f)
          sample_color.w = 0.f;
        else if(sample_color.w > 1.f)
          sample_color.w = 1.f;

        // store the output color
        sample_output.set_color(sample_color);

        return NV_IDX_PROG_OK;
    }
}; // class Volume_sample_program
