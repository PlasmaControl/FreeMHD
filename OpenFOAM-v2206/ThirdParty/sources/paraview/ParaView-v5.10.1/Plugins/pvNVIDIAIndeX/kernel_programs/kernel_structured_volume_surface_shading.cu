#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright 2021 NVIDIA Corporation. All rights reserved.
 *****************************************************************************/

// # XAC Kernel:
// ** Structured Volume Surface Shading **

// # Summary:
// Render the structured volume and optionally apply local surface shading at an user selected scalar threshold

NV_IDX_XAC_VERSION_1_0

// DON'T CHANGE THIS STRUCT !!!
// It maps the GUI parameters from the Custom Visual Element GUI to this kernel.
// floats maps {pfloat1, pfloat2, pfloat3, pfloat4} GUI Parameters.
// ints maps {pint1, pint2, pint3, pint4} GUI Parameters.
struct Custom_params
{
    float4 floats;  // floats array
    int4 ints;      // ints array
};

using namespace nv::index;
using namespace nv::index::xac;

class Volume_sample_program
{
    NV_IDX_VOLUME_SAMPLE_PROGRAM

    // sampling parameters
    const uint field_index      = 0u;       // volume field to use (default: 0)
    const float dh              = 1.0f;     // finite-difference stepsize (gradient approximation)

    // shading parameters
    const float diffuse_falloff = 1.0f;     // angular falloff for diffues lighting term
    const float shininess       = 100.0f;   // 'shininess' falloff parameter (lower is brighter)
    const float ambient_factor  = 0.1f;     // scaling of ambient term

    const float4 specular_color = make_float4(1.0f);

    const Custom_params*  m_custom_params;

public:
    NV_IDX_DEVICE_INLINE_MEMBER

    void initialize()
    {
         // maps Custom Visual Element GUI parameters to this kernel.
       m_custom_params = state.bind_parameter_buffer<Custom_params>(0);
    }

    NV_IDX_DEVICE_INLINE_MEMBER
    int execute(
        const Sample_info_self& sample_info,
              Sample_output&    sample_output)
    {
        // map "pint 1" GUI parameter as use shading option (true/false)
        // enable/disable shading effect
        const bool use_shading = (m_custom_params->ints.x >= 1);

        // map "pfloat 1" GUI parameter as min shading alpha
        // min alpha threshold for shading.
        float min_shade_alpha = m_custom_params->floats.x/100.f;
        if(min_shade_alpha < 0.f)
          min_shade_alpha = 0.f;

        // get current sample position
        const float3& sample_position = sample_info.sample_position_object_space;
        const auto& sample_context = sample_info.sample_context;

        // get reference to the current volume
        const auto& sparse_volume = state.self;

        // generate a volume sampler
        const auto sampler = sparse_volume.generate_sampler<float>(
                                                         field_index,
                                                         sample_context);

        // sample the volume at the current position
        const float sample_value = sampler.fetch_sample(sample_position);

        // sample the color value using the transfer function (colormap)
        const Colormap colormap = state.self.get_colormap();
        float4 sample_color = colormap.lookup(sample_value);

        // check if shading should be used
        if (use_shading && (sample_color.w > min_shade_alpha))
        {
            // make this sample full opaque
            sample_color.w = 1.f;

            // compute volume gradient using a trilinear filter
            const float3 gradient = xaclib::volume_gradient<Volume_filter_mode::TRILINEAR>(
                        sparse_volume,
                        sample_position,
                        field_index,
                        dh);

            // get shading parameters
            const float3& view_direction = sample_info.ray_direction;
            const float3 iso_normal = -normalize(gradient);

            // apply built-in headlight shading
            sample_color = xaclib::headlight_shading(
                        state.scene,
                        iso_normal,
                        view_direction,
                        sample_color,
                        specular_color,
                        diffuse_falloff,
                        shininess,
                        ambient_factor);
        }

        // store the output color
        sample_output.set_color(sample_color);

        return NV_IDX_PROG_OK;
    }
}; // class Volume_sample_program
