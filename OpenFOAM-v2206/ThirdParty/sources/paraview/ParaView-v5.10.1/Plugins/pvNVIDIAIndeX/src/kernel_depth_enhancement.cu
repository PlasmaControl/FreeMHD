#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// # RTC Kernel:
// ** Volume Depth Enhancement **

// # Summary:
// Apply a local front-to-back averaging along a predefined ray segment to darken samples in low
// alpha regions.

// Define the user-defined data structure
struct Depth_enhancement_params
{
  // common lighting params
  int light_mode;     // 0=headlight, 1=orbital
  float angle;        // 0 angle
  float elevation;    // pi/2 angle
  int max_dsteps;     // = 8 [GUI] number of additional samples
  float ash;          // 0.01f adaptive sampling threshold
  float screen_gamma; // 0.9f [GUI] gamma correction parameter

  // shading parameters [GUI / scene]
  float3 spec_color; // make_float3(1.0f) specular color
  float spec_fac;    // 0.2f specular factor (phong)
  float shininess;   // 50.0f shininess parameter (phong)
  float amb_fac;     // 0.4f ambient factor

  float shade_h;   // 0.5f [GUI] min alpha value for shading
  float min_alpha; // 0.001f min alpha for sampling (improves performance)

  float2 dummy;
};

using namespace nv::index::xac;
using namespace nv::index::xaclib;

class Volume_sample_program
{
  NV_IDX_VOLUME_SAMPLE_PROGRAM

public:
  const Depth_enhancement_params*
    m_depth_enhancement_params; // define variables to bind user-defined buffer to

public:
  NV_IDX_DEVICE_INLINE_MEMBER
  void initialize()
  {
    // Bind the contents of the buffer slot 0 to the variable
    m_depth_enhancement_params = state.bind_parameter_buffer<Depth_enhancement_params>(0);
  }

  NV_IDX_DEVICE_INLINE_MEMBER
  int execute(  const   Sample_info_self&   sample_info,
                        Sample_output&      sample_output)
  {
    const auto& volume = state.self;
    const float3& sample_position = sample_info.sample_position_object_space;
    const Colormap& colormap = volume.get_colormap();

    const auto  svol_sampler =
      volume.generate_sampler<float>(0u, sample_info.sample_context);

    // retrieve parameter buffer contents (fixed values in code definition)
    const float3 ray_dir = sample_info.ray_direction;

    float3 light_dir;
    if (m_depth_enhancement_params->light_mode == 0)
    {
      light_dir = ray_dir;
    }
    else
    {
      const float theta = m_depth_enhancement_params->angle;
      const float phi = m_depth_enhancement_params->elevation;
      light_dir = make_float3(sinf(phi) * cosf(theta), sinf(phi) * sinf(theta), cosf(phi));
    }

    // sample volume
    const float rh = state.self.get_sample_distance() * 2.0f; // ray sampling difference
    const float3 p0 = sample_position;
    const float3 p1 = sample_position + (ray_dir * rh);

    const float vs_p0 = svol_sampler.fetch_sample(p0);
    const float vs_p1 = svol_sampler.fetch_sample(p1);

    const float vs_min = min(vs_p0, vs_p1);
    const float vs_max = max(vs_p0, vs_p1);

    // set adaptive sampling
    int d_steps = m_depth_enhancement_params->max_dsteps;

    if (m_depth_enhancement_params->ash > 0.0f)
    {
      int((vs_max - vs_min) /
        fabsf(m_depth_enhancement_params->ash)); // get number of additional samples
      d_steps = min(d_steps, m_depth_enhancement_params->max_dsteps);
    }

    // sample once
    const float4 sample_color = colormap.lookup(vs_p0);
    sample_output.color = make_float4(sample_color.x, sample_color.y, sample_color.z, sample_color.w);

    if (sample_output.color.w < m_depth_enhancement_params->min_alpha)
      return NV_IDX_PROG_DISCARD_SAMPLE;

    if (d_steps > 1)
    {
      // init result color
      float4 result = make_float4(0.0f);

      // iterate steps
      for (int ahc = 0; ahc < d_steps; ahc++)
      {
        // get step size
        const float rt = float(ahc) / float(d_steps);
        const float3 pi = (1.0f - rt) * p0 + rt * p1;
        const float vs_pi = svol_sampler.fetch_sample(pi);
        const float4 cur_col = colormap.lookup(vs_pi);

        // front-to-back blending
        const float omda = (1.0f - result.w);

        result.x += omda * cur_col.x * cur_col.w;
        result.y += omda * cur_col.y * cur_col.w;
        result.z += omda * cur_col.z * cur_col.w;
        result.w += omda * cur_col.w;
      }

      // assign result color
      sample_output.color = result;
    }
    // local shading
    if (sample_output.color.w > m_depth_enhancement_params->shade_h)
    {
      // get gradient normal
      const float3 vs_grad = volume_gradient<Volume_filter_mode::TRILINEAR>(volume, sample_position);
      const float3 iso_normal = -normalize(vs_grad);               // get isosurface normal

      // set up lighting parameters
      const float3 view_dir = sample_info.ray_direction;

      const float3 diffuse_color = make_float3(sample_output.color);

      const float diff_amnt = fabsf(dot(light_dir, iso_normal)); // two sided shading
      float spec_amnt = 0.0f;

      if (diff_amnt > 0.0f)
      {
        // this is blinn phong
        const float3 H = normalize(light_dir + view_dir);
        const float NH = fabsf(dot(H, iso_normal)); // two sided shading
        spec_amnt = powf(NH, m_depth_enhancement_params->shininess);
      }

      // compute final color (RGB)
      const float3 color_linear =
        diffuse_color * (m_depth_enhancement_params->amb_fac + diff_amnt) +
        m_depth_enhancement_params->spec_color * (spec_amnt * m_depth_enhancement_params->spec_fac);

      // apply gamma correction (assume ambient_color, diffuse_color and spec_color
      // have been linearized, i.e. have no gamma correction in them)
      sample_output.color.x = powf(color_linear.x, float(1.0f / m_depth_enhancement_params->screen_gamma));
      sample_output.color.y = powf(color_linear.y, float(1.0f / m_depth_enhancement_params->screen_gamma));
      sample_output.color.z = powf(color_linear.z, float(1.0f / m_depth_enhancement_params->screen_gamma));

      // apply build in gamma function
      // sample_output.color = gamma_correct(sample_output.color, m_depth_enhancement_params->screen_gamma);

      // clamp result color
      sample_output.color = clamp(sample_output.color, 0.0f, 1.0f);
    }

    return NV_IDX_PROG_OK;
  }
}; // class Volume_sample_program
