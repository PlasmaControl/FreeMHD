#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
* * Redistributions of source code must retain the above copyright
*  notice, this list of conditions and the following disclaimer.
* * Redistributions in binary form must reproduce the above copyright
*  notice, this list of conditions and the following disclaimer in the
*  documentation and/or other materials provided with the distribution.
* * Neither the name of NVIDIA CORPORATION nor the names of its
*  contributors may be used to endorse or promote products derived
*  from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
// # RTC Kernel:
// ** Volume Depth Enhancement **
// # Summary:
// Apply a local front-to-back averaging along a predefined ray segment to darken samples in low
// alpha regions.
// Define the user-defined data structure

NV_IDX_XAC_VERSION_1_0

struct Depth_enhancement_params
{
    float rh;
    // common lighting params
    int light_mode;   // 0=headlight, 1=orbital
    float angle;    // 0 angle
    float elevation;  // pi/2 angle
    int max_dsteps;   // = 8 [GUI] number of additional samples
    float screen_gamma; // 0.9f [GUI] gamma correction parameter

    // shading parameters [GUI / scene]
    float3 spec_color; // make_float3(1.0f) specular color
    float spec_fac;  // 0.2f specular factor (phong)
    float shininess;  // 50.0f shininess parameter (phong)
    float amb_fac;   // 0.4f ambient factor
};

using namespace nv::index;
using namespace nv::index::xac;

class Volume_sample_program
{
    NV_IDX_VOLUME_SAMPLE_PROGRAM

    const uint field_index      = 0u;
    const float min_grad_length = 0.001f;
    const float shade_h         = 0.5f; // min alpha value for shading
    const float min_alpha       = 0.001f; // min alpha for sampling

public:
    const Depth_enhancement_params*
    m_depth_enhancement_params; // define variables to bind user-defined buffer to

public:
    NV_IDX_DEVICE_INLINE_MEMBER
    void initialize()
    {
        // Bind the contents of the buffer slot 0 to the variable
        m_depth_enhancement_params = state.bind_parameter_buffer<Depth_enhancement_params>(0);
    }
    NV_IDX_DEVICE_INLINE_MEMBER
    int execute(
        const   Sample_info_self&  sample_info,
                Sample_output&   sample_output)
    {

        // retrieve ray intersection properties
        const float3& scene_position = sample_info.scene_position;
        const float3& ray_dir = sample_info.ray_direction;

        // get sparse volume reference
        const auto& cell_info = sample_info.sample_cell_info;
        const auto& irregular_volume = state.self;

        // get the associated colormap
        const Colormap colormap = irregular_volume.get_colormap();

        // retrieve user parameter buffer contents
        float3 light_dir = ray_dir;

        if (m_depth_enhancement_params->light_mode != 0)
        {
            const float theta = m_depth_enhancement_params->angle;
            const float phi = m_depth_enhancement_params->elevation;
            light_dir = make_float3(sinf(phi) * cosf(theta), sinf(phi) * sinf(theta), cosf(phi));
        }

        // sample volume along the ray
        const float rh = m_depth_enhancement_params->rh;
        const float3 p0 = scene_position;
        const float3 p1 = p0 + (ray_dir * rh);
        const float3 v01 = (ray_dir * rh);

        const float vs_p0 = irregular_volume.fetch_attribute<float>(field_index, cell_info);

        // init sample color sample
        float4 sample_color  = colormap.lookup(vs_p0);
        sample_output.set_color(sample_color);

        // stop computation if opacity is below threshold (improves performance)
        if (sample_color.w < min_alpha)
        {
            return NV_IDX_PROG_OK;
        }

        // check the number of steps to take along the ray
        if (m_depth_enhancement_params->max_dsteps >= 2)
        {
            // init result color
            float4 acc_color = make_float4(
              sample_color.x * sample_color.w,
              sample_color.y * sample_color.w,
              sample_color.z * sample_color.w,
              sample_color.w);

            // iterate steps
            for (int ahc=1; ahc < m_depth_enhancement_params->max_dsteps; ahc++)
            {
                // get step size
                const float rt = float(ahc) / float(m_depth_enhancement_params->max_dsteps);
                const float3 pi = p0 + (p1 - p0)*rt;
                const float3 v0i = pi - p0;

                // sample current position
                const float vs_pi = irregular_volume.fetch_attribute_offset<float>(field_index, cell_info, v0i);
                const float4 cur_col  = colormap.lookup(vs_pi);

                // front-to-back blending
                const float omda = (1.0f - acc_color.w);

                acc_color.x += omda * cur_col.x * cur_col.w;
                acc_color.y += omda * cur_col.y * cur_col.w;
                acc_color.z += omda * cur_col.z * cur_col.w;
                acc_color.w += omda * cur_col.w;
            }

            // assign result color
            sample_color = acc_color;
            sample_output.set_color(sample_color);
        }

        // check if local shading has to be applied
        if (sample_color.w > shade_h)
        {
            // get the volume gradient
            const float3 vs_grad = get_gradient_3n(sample_info, vs_p0, rh);

            // check gradient length
            if (length(vs_grad) < min_grad_length)
              return NV_IDX_PROG_OK;

            // get isosurface normal
            const float3 iso_normal = -normalize(vs_grad);

            // set ambient & diffuse color
            const float3 diffuse_color = make_float3(sample_color);

            // init shading parameters
            const float lambertian = fabsf(dot(light_dir,iso_normal));
            float spec_fac = 0.0f;

            // check normal direction (two-sided shading)
            if(lambertian > 0.0f)
            {
                // this is blinn phong
                const float3 half_dir = normalize(light_dir + ray_dir);
                const float spec_angle = fabsf(dot(half_dir, iso_normal));

                spec_fac = powf(spec_angle, m_depth_enhancement_params->shininess);
            }

            // compute final shaded color (RGB)
            float4 color_linear = make_float4(diffuse_color * (m_depth_enhancement_params->amb_fac + lambertian) + m_depth_enhancement_params->spec_color * (spec_fac * m_depth_enhancement_params->spec_fac));

            // apply gamma correction
            color_linear = xaclib::gamma_correct(color_linear, m_depth_enhancement_params->screen_gamma);
            color_linear.w = sample_color.w;
            color_linear = clamp(color_linear, 0.0f, 1.0f);

            // clamp result color
            sample_output.set_color(color_linear);
        }

        return NV_IDX_PROG_OK;
    }

    // compute gradient for irregular volume
    NV_IDX_DEVICE_INLINE_MEMBER
    float3 get_gradient_3n(const Sample_info_self&  sample_info, float vs_c, float rh)
    {
        const float dh = 0.5f*rh;
        const float dh_inv = 1.0f/rh;
        const auto& cell_info = sample_info.sample_cell_info;
        const auto& ivol = state.self;

        // get spatial sample points for each dimensions
        // const float vs_c = ivol.fetch_attribute<float>(0u, cell_info);

        const float vs_dx_p = ivol.fetch_attribute_offset<float>(0u, cell_info, make_float3( dh, 0, 0));
        const float vs_dy_p = ivol.fetch_attribute_offset<float>(0u, cell_info, make_float3(  0, dh, 0));
        const float vs_dz_p = ivol.fetch_attribute_offset<float>(0u, cell_info, make_float3(  0, 0, dh));

        // get R3 gradient vector
        return make_float3(
            (vs_dx_p - vs_c) * dh_inv,
            (vs_dy_p - vs_c) * dh_inv,
            (vs_dz_p - vs_c) * dh_inv);

    }
}; // class Volume_sample_program
