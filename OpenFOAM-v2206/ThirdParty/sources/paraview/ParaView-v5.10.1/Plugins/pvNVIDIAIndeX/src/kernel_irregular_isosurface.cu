#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// # RTC Kernel:
// ** Volume Isosurface Raycaster **

// # Summary:
// Compute the intersection of an isosurface with a user-controlled iso value along the current
// ray segment and shade using a fixed Blinn-Phong model.

// Define the user-defined data structure
struct Isosurface_params
{
  float rh;        // raycast step size
  // Common lighting parameter.
  int light_mode;  // 0=headlight, 1=orbital
  float angle;     // 0.0 angle
  float elevation; // 0.0 elevation

  float iso_min;   // 0.5, iso value in %
  float iso_max;   // 0.5, iso value in %
  int fill_up;     // 1
  int use_shading; // 1, use local phong-blinn model

  float spec_fac;  // 1.0f, specular level (phong)
  float shininess; // 50.0f, shininess parameter (phong)
  float amb_fac;   // 0.2f, ambient factor

  float3 spec_color; // make_float3(1.0f), specular color

  float dummy[2]; // for memory 16 bytes alignment
};

using namespace nv::index;
using namespace nv::index::xac;

class Volume_sample_program
{
  NV_IDX_VOLUME_SAMPLE_PROGRAM

  const uint field_index  = 0u;
  const float min_alpha   = 0.05f; // finite difference
  const float diff_exp    = 2.0f; // diffuse falloff (like edge enhance)

public:
  const Isosurface_params* m_isosurface_params; // define variables to bind user-defined buffer to

public:
  NV_IDX_DEVICE_INLINE_MEMBER
  void initialize()
  {
    // Bind the contents of the buffer slot 0 to the variable
    m_isosurface_params = state.bind_parameter_buffer<Isosurface_params>(0);
  }

  NV_IDX_DEVICE_INLINE_MEMBER
  int execute(const Sample_info_self&   sample_info,
                    Sample_output&      sample_output)
  {
    const auto& cell_info = sample_info.sample_cell_info;
    const auto& volume = state.self;
    const float3 ray_dir = sample_info.ray_direction;
    const Colormap& colormap = volume.get_colormap();

    // retrieve parameter buffer contents (fixed values in code definition)
    float iso_min = (1.0f - m_isosurface_params->iso_min) * colormap.get_domain_min() +
      m_isosurface_params->iso_min * colormap.get_domain_max();
    float iso_max = (1.0f - m_isosurface_params->iso_max) * colormap.get_domain_min() +
      m_isosurface_params->iso_max * colormap.get_domain_max();

    if (iso_max < iso_min)
      return NV_IDX_PROG_DISCARD_SAMPLE;

    // sample volume and colormap
    const float volume_sample = volume.fetch_attribute<float>(field_index, cell_info);

    // get spatial sample points for each dimensions
    const float rh = m_isosurface_params->rh;
    const float dh = 0.1f*rh;
    const float vs_dr_p = volume.fetch_attribute_offset<float>(field_index, cell_info, ray_dir * rh);
    const float vs_dr_n = volume.fetch_attribute_offset<float>(field_index, cell_info, - ray_dir * rh);

    // sum up threshold exceeds in both directions for iso_min
    float sum_over_min = float((vs_dr_p > iso_min) + (vs_dr_n > iso_min) + (volume_sample > iso_min));
    float sum_under_min = float((vs_dr_p < iso_min) + (vs_dr_n < iso_min) + (volume_sample < iso_min));

    // check for iso_min intersections
    if (sum_over_min > 0 && sum_under_min > 0)
    {
      // sample color
      const float4 sample_color = colormap.lookup(iso_min); // use isovalue color

      // check if to skip sample
      if (sample_color.w < min_alpha)
        return NV_IDX_PROG_DISCARD_SAMPLE;

      // valid intersection found
      if (m_isosurface_params->use_shading)
      {
        const float3 iso_normal =
          -normalize(xaclib::volume_gradient(volume, cell_info, dh)); // get isosurface normal
        sample_output.color = blinn_shader(iso_normal, sample_color, ray_dir);
      }
      else
      {
        // use sample color
        sample_output.color = sample_color;
        sample_output.color.w = 1.0f;
      }

      return NV_IDX_PROG_OK;
    }
    else if(sum_over_min >= 3)
    {
      // sum up threshold exceeds in both directions for iso_max
      float sum_over_max = float((vs_dr_p > iso_max) + (vs_dr_n > iso_max) + (volume_sample > iso_max));
      float sum_under_max = float((vs_dr_p < iso_max) + (vs_dr_n < iso_max) + (volume_sample < iso_max));

      // check for iso_max intersections
      if (sum_over_max > 0 && sum_under_max > 0)
      {
        // sample color
        const float4 sample_color = colormap.lookup(iso_max); // use isovalue color

        // check if to skip sample
        if (sample_color.w < min_alpha)
          return NV_IDX_PROG_DISCARD_SAMPLE;

        // valid intersection found
        if (m_isosurface_params->use_shading)
        {
          const float3 iso_normal =
            -normalize(xaclib::volume_gradient(volume, cell_info, dh)); // get isosurface normal
          sample_output.color = blinn_shader(iso_normal, sample_color, ray_dir);
        }
        else
        {
          // use sample color
          sample_output.color = sample_color;
          sample_output.color.w = 1.0f;
        }

        return NV_IDX_PROG_OK;
      }
      else if(sum_under_max >=3 && m_isosurface_params->fill_up > 0)
      {
        if(m_isosurface_params->fill_up == 1)
        {
          // use iso_min color
          sample_output.color = colormap.lookup(iso_min);
          sample_output.color.w = 1.0f;
          return NV_IDX_PROG_OK;
        }
        else
        {
          // use sample color
          sample_output.color = colormap.lookup(volume_sample);
          return NV_IDX_PROG_OK;
        }
      }
      else
      {
        // no isosurface intersection
        return NV_IDX_PROG_DISCARD_SAMPLE;
      }
    }
    else
    {
      // no isosurface intersection
      return NV_IDX_PROG_DISCARD_SAMPLE;
    }
  }

  NV_IDX_DEVICE_INLINE_MEMBER
  float4 blinn_shader(const float3& normal, const float4& sample_color, const float3& ray_dir)
  {
    const float3 diffuse_color = make_float3(sample_color);

    float NL, NH;
    if (m_isosurface_params->light_mode == 0)
    {
      NL = NH = fabsf(dot(ray_dir, normal));
    }
    else
    {
      const float theta = m_isosurface_params->angle;
      const float phi = m_isosurface_params->elevation;
      float3 light_dir = make_float3(sinf(phi) * cosf(theta), sinf(phi) * sinf(theta), cosf(phi));

      NL = fabsf(dot(light_dir, normal));
      float3 H = normalize(light_dir + ray_dir);
      NH = fabsf(dot(H, normal));
    }

    const float diff_amnt = powf(NL, diff_exp);
    const float spec_amnt = powf(NH, m_isosurface_params->shininess);

    // compute final color (RGB)
    const float3 shade_color = diffuse_color * (m_isosurface_params->amb_fac + diff_amnt) +
      m_isosurface_params->spec_color * (m_isosurface_params->spec_fac * spec_amnt);

    return clamp(
      make_float4(shade_color.x, shade_color.y, shade_color.z, sample_color.w), 0.0f, 1.0f);
  }
}; // class Volume_sample_program
