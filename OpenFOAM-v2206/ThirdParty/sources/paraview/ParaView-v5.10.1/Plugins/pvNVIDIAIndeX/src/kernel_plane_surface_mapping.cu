#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright 2021 NVIDIA Corporation. All rights reserved.
 *****************************************************************************/

#define SLOT_VOLUME   1
#define SLOT_COLORMAP 2

class Surface_sample_program
{
    NV_IDX_SURFACE_SAMPLE_PROGRAM
public:
    NV_IDX_DEVICE_INLINE_MEMBER
    void initialize() {}

    NV_IDX_DEVICE_INLINE_MEMBER
    int execute(
        const Sample_info_self& sample_info,        // read-only
              Sample_output&    sample_output)      // write-only
    {
        using namespace nv::index;

        if (!state.scene.is_valid_element<xac::Sparse_volume>(SLOT_VOLUME)) {
            return NV_IDX_PROG_DISCARD_SAMPLE;
        }

        const auto      svol      = state.scene.access<xac::Sparse_volume>(SLOT_VOLUME);
        const float3    svol_spos = transform_point(svol.get_scene_to_object_transform(), sample_info.scene_position);

        const float3    svol_spos_offset = svol_spos + make_float3(0.0f, 0.0f, 0.0f);

        const auto svol_sampler = svol.generate_sampler<float,
                                                        xac::Volume_filter_mode::TRILINEAR,
                                                        xac::Volume_classification_mode::POST_INTERPOLATION>(
                                                            0u);

        // Get a sample at a shifted position to make sure access happens outside of boundary
        // of local sparse volume.

        const auto colormap = state.scene.access<xac::Colormap>(SLOT_COLORMAP);

        const float4 svol_sample = svol_sampler.fetch_sample_classify(svol_spos_offset, colormap);

        sample_output.set_color(svol_sample);

        return NV_IDX_PROG_OK;
    }
};
