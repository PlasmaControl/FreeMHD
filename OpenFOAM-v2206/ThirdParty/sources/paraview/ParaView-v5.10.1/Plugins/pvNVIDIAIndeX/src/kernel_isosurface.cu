#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// # RTC Kernel:
// ** Volume Isosurface Raycaster **

// # Summary:
// Compute the intersection of an isosurface with a user-controlled iso value along the current
// ray segment and shade using a fixed Blinn-Phong model.

// Define the user-defined data structure
struct Isosurface_params
{
  // common lighting params
  int light_mode;  // 0=headlight, 1=orbital
  float angle;     // 0.0 angle
  float elevation; // 0.0 elevation

  float iso_min;   // 0.5, iso value in %
  float iso_max;   // 0.5, iso value in %
  int fill_up;     // 1
  int use_shading; // 1, use local phong-blinn model
  float min_alpha; // 0.05, finite difference

  float spec_fac;  // 1.0f, specular level (phong)
  float shininess; // 50.0f, shininess parameter (phong)
  float amb_fac;   // 0.2f, ambient factor
  float diff_exp;  // 2.0f, diffuse falloff (like edge enhance)

  int show_grid;  // 0, show normal grid
  int ng_num;     // 16
  float ng_width; // 0.01f

  float3 spec_color; // make_float3(1.0f), specular color

  float2 dummy; // for memory 16 bytes alignment
};

using namespace nv::index::xac;
using namespace nv::index::xaclib;

class Volume_sample_program
{
  NV_IDX_VOLUME_SAMPLE_PROGRAM

public:
  const Isosurface_params* m_isosurface_params; // define variables to bind user-defined buffer to

public:
  NV_IDX_DEVICE_INLINE_MEMBER
  void initialize()
  {
    // Bind the contents of the buffer slot 0 to the variable
    m_isosurface_params = state.bind_parameter_buffer<Isosurface_params>(0);
  }

  NV_IDX_DEVICE_INLINE_MEMBER
  int execute(const Sample_info_self&   sample_info,
                    Sample_output&      sample_output)
  {
    const auto& volume = state.self;
    const float3& sample_position = sample_info.sample_position_object_space;
    const float3 ray_dir = sample_info.ray_direction;
    const Colormap& colormap = volume.get_colormap();

    // retrieve parameter buffer contents (fixed values in code definition)
    float iso_min = (1.0f - m_isosurface_params->iso_min) * colormap.get_domain_min() +
      m_isosurface_params->iso_min * colormap.get_domain_max();
    float iso_max = (1.0f - m_isosurface_params->iso_max) * colormap.get_domain_min() +
      m_isosurface_params->iso_max * colormap.get_domain_max();

    if (iso_max < iso_min)
      return NV_IDX_PROG_DISCARD_SAMPLE;

    // sample volume and colormap
    const auto  svol_sampler = volume.generate_sampler<float>(
                                                        0u,
                                                        sample_info.sample_context);

    const float volume_sample = svol_sampler.fetch_sample(sample_position);

    // get spatial sample points for each dimensions
    const float rh = volume.get_sample_distance(); // ray sampling difference
    const float vs_dr_p = svol_sampler.fetch_sample(sample_position + ray_dir * rh);
    const float vs_dr_n = svol_sampler.fetch_sample(sample_position - ray_dir * rh);

    // sum up threshold exceeds in both directions for iso_min
    float sum_over_min = float((vs_dr_p > iso_min) + (vs_dr_n > iso_min) + (volume_sample > iso_min));
    float sum_under_min = float((vs_dr_p < iso_min) + (vs_dr_n < iso_min) + (volume_sample < iso_min));

    // check for iso_min intersections
    if (sum_over_min > 0 && sum_under_min > 0)
    {
      // sample color
      const float4 sample_color = colormap.lookup(iso_min); // use isovalue color

      // check if to skip sample
      if (sample_color.w < m_isosurface_params->min_alpha)
      {
        return NV_IDX_PROG_DISCARD_SAMPLE;
      }
      else
      {
        // valid intersection found
        if (m_isosurface_params->use_shading)
        {
          // get isosurface normal
          const float3 iso_normal =
            -normalize(volume_gradient<Volume_filter_mode::TRILINEAR>(volume, sample_position));
          sample_output.color = blinn_shader(iso_normal, sample_color, ray_dir);
        }
        else
        {
          // use sample color
          sample_output.color = sample_color;
          sample_output.color.w = 1.0f;
        }

        return NV_IDX_PROG_OK;
      }
    }
    else if(sum_over_min >= 3)
    {
      // sum up threshold exceeds in both directions for iso_max
      float sum_over_max = float((vs_dr_p > iso_max) + (vs_dr_n > iso_max) + (volume_sample > iso_max));
      float sum_under_max = float((vs_dr_p < iso_max) + (vs_dr_n < iso_max) + (volume_sample < iso_max));

      // check for iso_max intersections
      if (sum_over_max > 0 && sum_under_max > 0)
      {
        // sample color
        const float4 sample_color = colormap.lookup(iso_max); // use isovalue color

        // check if to skip sample
        if (sample_color.w < m_isosurface_params->min_alpha)
          return NV_IDX_PROG_DISCARD_SAMPLE;

        // valid intersection found
        if (m_isosurface_params->use_shading)
        {
          // get isosurface normal
          const float3 iso_normal =
            -normalize(volume_gradient<Volume_filter_mode::TRILINEAR>(volume, sample_position));
          sample_output.color = blinn_shader(iso_normal, sample_color, ray_dir);
        }
        else
        {
          // use sample color
          sample_output.color = sample_color;
          sample_output.color.w = 1.0f;
        }

        return NV_IDX_PROG_OK;
      }
      else if(sum_under_max >=3 && m_isosurface_params->fill_up > 0)
      {
        if(m_isosurface_params->fill_up == 1)
        {
          // use iso_min color
          sample_output.color = colormap.lookup(iso_min);
          sample_output.color.w = 1.0f;
          return NV_IDX_PROG_OK;
        }
        else
        {
          // use sample color
          sample_output.color = colormap.lookup(volume_sample);
          return NV_IDX_PROG_OK;
        }
      }
      else
      {
        // no isosurface intersection
        return NV_IDX_PROG_DISCARD_SAMPLE;
      }
    }
    else
    {
      // no isosurface intersection
      return NV_IDX_PROG_DISCARD_SAMPLE;
    }
  }

  NV_IDX_DEVICE_INLINE_MEMBER
  float4 blinn_shader(const float3& normal, const float4& sample_color, const float3& ray_dir)
  {
    const float3 diffuse_color = make_float3(sample_color);

    float NL, NH;
    if (m_isosurface_params->light_mode == 0)
    {
      NL = NH = fabsf(dot(ray_dir, normal));
    }
    else
    {
      const float theta = m_isosurface_params->angle;
      const float phi = m_isosurface_params->elevation;
      float3 light_dir = make_float3(sinf(phi) * cosf(theta), sinf(phi) * sinf(theta), cosf(phi));

      NL = fabsf(dot(light_dir, normal));
      float3 H = normalize(light_dir + ray_dir);
      NH = fabsf(dot(H, normal));
    }

    const float diff_amnt = powf(NL, m_isosurface_params->diff_exp);
    const float spec_amnt = powf(NH, m_isosurface_params->shininess);

    // compute final color (RGB)
    const float3 shade_color = diffuse_color * (m_isosurface_params->amb_fac + diff_amnt) +
      m_isosurface_params->spec_color * (m_isosurface_params->spec_fac * spec_amnt);

    return clamp(
      make_float4(shade_color.x, shade_color.y, shade_color.z, sample_color.w), 0.0f, 1.0f);
  }
}; // class Volume_sample_program
