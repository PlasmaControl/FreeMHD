#include "hip/hip_runtime.h"
/* Copyright 2021 NVIDIA Corporation. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// # RTC Kernel:
// ** Volume Edge Enhancement **

// # Summary:
// Compute the colormap alpha gradient in the current volume and darken samples with high gradient
// magnitude a a user-defined rate.

// Define the user-defined data structure
struct Edge_enhancement_params
{
  float sample_range; // [10, 10] 1.0 sample range
  float rh;           // 1.0f ray sampling difference
  int stp_num;        // 6 [GUI] additional samples along ray
  float min_alpha;    // 0.2f minimum alpha for sampling (improves performance)
};

using namespace nv::index::xac;
using namespace nv::index::xaclib;

class Volume_sample_program
{
  NV_IDX_VOLUME_SAMPLE_PROGRAM

public:
  const Edge_enhancement_params*
    m_edge_enhancement_params; // define variables to bind user-defined buffer to

public:
  NV_IDX_DEVICE_INLINE_MEMBER
  void initialize()
  {
    // Bind the contents of the buffer slot 0 to the variable
    m_edge_enhancement_params = state.bind_parameter_buffer<Edge_enhancement_params>(0);
  }

  NV_IDX_DEVICE_INLINE_MEMBER
  int execute(const Sample_info_self&   sample_info,
                    Sample_output&      sample_output)
  {
    const auto& volume = state.self;
    const float3& ps = sample_info.sample_position_object_space;
    const Colormap& colormap = volume.get_colormap();

    const auto  svol_sampler =
      volume.generate_sampler<float>(0u, sample_info.sample_context);


    // sample volume
    const float3 ray_dir = sample_info.ray_direction;
    const float3 p0 = ps - (ray_dir * (m_edge_enhancement_params->rh *
                                       m_edge_enhancement_params->sample_range));
    const float3 p2 = ps + (ray_dir * (m_edge_enhancement_params->rh *
                                       m_edge_enhancement_params->sample_range));

    const float vs_p0 = svol_sampler.fetch_sample(p0);
    const float vs_ps = svol_sampler.fetch_sample(ps);
    const float vs_p2 = svol_sampler.fetch_sample(p2);

    const float4 ps_color = colormap.lookup(vs_ps);
    const float ps_alpha = ps_color.w;
    const int min_hits = m_edge_enhancement_params->stp_num;
    const int num_steps = min_hits * 2;

    if (ps_alpha < m_edge_enhancement_params->min_alpha)
    {
      // sample below given alpha threshold
      // return NV_IDX_PROG_DISCARD_SAMPLE;
    }

    if ((num_steps > 0) && (ps_alpha > m_edge_enhancement_params->min_alpha))
    {
      // init result color
      int sum_over_b = 0;
      int sum_under_b = 0;
      int sum_over_a = 0;
      int sum_under_a = 0;

      // iterate steps
      for (int sc = 0; sc <= num_steps; sc++)
      {
        // get step size
        const float rt = float(sc) / float(num_steps);
        const float3 pi_b = (1.0f - rt) * p0 + rt * ps;
        const float3 pi_a = (1.0f - rt) * ps + rt * p2;

        const float vs_pi_b = svol_sampler.fetch_sample(pi_b);
        const float vs_pi_a = svol_sampler.fetch_sample(pi_a);

        const float4 pia_color = colormap.lookup(vs_pi_b);
        const float4 pib_color = colormap.lookup(vs_pi_a);

        // check by alpha
        if (pib_color.w > ps_alpha)
          sum_over_b += 1;
        if (pib_color.w < ps_alpha)
          sum_under_b += 1;
        if (pia_color.w > ps_alpha)
          sum_over_a += 1;
        if (pia_color.w < ps_alpha)
          sum_under_a += 1;
      }

      // check if all are larger
      // if (sum_over > 1 && sum_under == 0)
      //    sample_output.color = make_float4(ps_color.x, ps_color.y, ps_color.z, ps_color.w);

      // check numer of samples below / above reference
      if (((sum_under_b > min_hits) && (sum_under_a > min_hits)) ||
        ((sum_over_b > min_hits) && (sum_over_a > min_hits)))
      {
        sample_output.color = ps_color / float(sum_under_b + sum_under_a - 2 * min_hits + 1);
        sample_output.color.w = ps_color.w;

        return NV_IDX_PROG_OK;
      }
      else
      {
        sample_output.color = ps_color;

        // return NV_IDX_PROG_DISCARD_SAMPLE;
      }
    }
    else
    {
      sample_output.color = ps_color;
    }

    return NV_IDX_PROG_OK;
  }
}; // class Volume_sample_program
